#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#define SIZE 100
#define threads 32

using namespace std;

__global__ void addTwoArrays(int *d_v1, int *d_v2, int *d_vsum, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < n)
    {
        d_vsum[index] = d_v1[index] + d_v2[index];
    }
}

int main()
{
    int *v1;
    int *v2;
    int *vsum;

    int *d_v1;
    int *d_v2;
    int *d_vsum;

    size_t bytes = SIZE * sizeof(int);

    v1 = (int *)malloc(bytes);
    v2 = (int *)malloc(bytes);
    vsum = (int *)malloc(bytes);

    hipMalloc(&d_v1, bytes);
    hipMalloc(&d_v2, bytes);
    hipMalloc(&d_vsum, bytes);

    for (int i = 0; i < SIZE; i++)
    {
        v1[i] = i;
        v2[i] = SIZE;
    }

    hipMemcpy(d_v1, v1, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_v2, v2, bytes, hipMemcpyHostToDevice);
    addTwoArrays<<<(SIZE + 127) / 128, 128>>>(d_v1, d_v2, d_vsum, SIZE);
    hipMemcpy(vsum, d_vsum, bytes, hipMemcpyDeviceToHost);

    cout << "rez: ";

    for (int i = 0; i < SIZE; i++)
    {
        cout << v1[i] << " " << v2[i] << " " << vsum[i] << endl;
    }

    cout << endl;

    hipFree(d_v1);
    hipFree(d_v2);
    hipFree(d_vsum);
    free(v1);
    free(v2);
    free(vsum);

    hipDeviceReset();
    return 0;
}