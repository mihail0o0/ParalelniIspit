#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#define SIZE 1024

using namespace std;

__global__ void multiplyMatrices(int *d_v1, int *d_v2, int *d_vsum, int n)
{
}

int main()
{
    size_t bytes = SIZE * SIZE * sizeof(int);

    int *a, *b, *rez;
    int *d_a, *d_b, *d_rez;

    a = (int *)malloc(bytes);
    b = (int *)malloc(bytes);
    rez = (int *)malloc(bytes);

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_rez, bytes);

    for (int i = 0; i < SIZE; i++)
    {
        for (int j = 0; j < SIZE; j++)
        {
            int index = i * SIZE + j;
            a[index] = index;
            b[index] = index;
        }
    }

    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, bytes, hipMemcpyHostToDevice);

    int blockSize = 16;
    int gridSize = (SIZE + blockSize - 1 / blockSize);

    dim3 grids(gridSize, gridSize);
    dim3 threads(blockSize, blockSize);

    multiplyMatrices<<<grids, threads>>>(d_a, d_b, d_rez, SIZE * SIZE);

    return 0;
}